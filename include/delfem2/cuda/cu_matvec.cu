#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include "hip/hip_runtime.h"

#include "cu_matvec.h"

namespace dfm2 = delfem2;

// -------------------------------------------------------------------

__device__
void atomicMaxFloat(float * const address, const float value)
{
  if ( *address >= value ) { return; }
  int * const address_as_i = (int *)address;
  int old = * address_as_i, assumed;
  do {
    assumed = old;
    if (__int_as_float(assumed) >= value) { break; }
    old = atomicCAS(address_as_i, assumed, __float_as_int(value));
  } while (assumed != old);
}

__device__
void atomicMinFloat(float * const address, const float value)
{
  if ( *address <= value ) { return; }
  int * const address_as_i = (int *)address;
  int old = * address_as_i, assumed;
  do {
    assumed = old;
    if (__int_as_float(assumed) <= value) { break; }
    old = atomicCAS(address_as_i, assumed, __float_as_int(value));
  } while (assumed != old);
}


// -------------------------------------------------------------------

__global__
void dfm2::cuda::kernel_VecScale(
    float *out,
    const float *in,
    float scale,
    const int n)
{
  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) { return; }
  out[i] = in[i] * scale;
}

void dfm2::cuda::cuda_VecScale(
    float *hOut,
    const float *hIn,
    float scale,
    const int n)
{
  float *dOut; hipMalloc((void**)&dOut, sizeof(float)*n);
  float *dIn;  hipMalloc((void**)&dIn,  sizeof(float)*n);
  hipMemcpy(dIn, hIn, sizeof(float)*n, hipMemcpyHostToDevice);

  const unsigned int tpb = 64;
  const unsigned int nblk = (unsigned int)((n-1)/tpb+1);
  kernel_VecScale<<<nblk, tpb>>>(dOut, dIn, scale, n);
  hipDeviceSynchronize();

  hipMemcpy(hOut, dOut, n * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(dOut);
  hipFree(dIn);
}


// -------------------------------------------------------------------

/**
 * @brief dot product of two vectors
 */
__global__
void dfm2::cuda::kernel_Dot_TPB64(
    float* d_res,
    const float* d_A,
    const float* d_B,
    int n)
{
  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if( idx >= n ){ return; }

  const int s_idx = threadIdx.x;

  const unsigned int TPB = 64;
  __shared__ float s_prod[TPB];
  s_prod[s_idx] = d_A[idx]*d_B[idx];
  __syncthreads();

  if( s_idx == 0 ) {
    float blockSum = 0;
    for(int j=0;j<blockDim.x;++j){
      blockSum += s_prod[j];
    }
    atomicAdd(d_res, blockSum);
  }
}


float dfm2::cuda::cuda_Dot(
    const float* h_A,
    const float* h_B,
    unsigned int n)
{
  float *d_A, *d_B, *d_res;
  hipMalloc((void **) &d_A, sizeof(float) * n);
  hipMalloc((void **) &d_B, sizeof(float) * n);
  hipMalloc((void **) &d_res, sizeof(float));

  hipMemset((void **) &d_res, 0.f, sizeof(float));
  hipMemcpy(d_A, h_A, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, sizeof(float) * n, hipMemcpyHostToDevice);

  const unsigned int BLOCK = 64;
  dim3 grid(n / BLOCK);
  dim3 block(BLOCK);

  kernel_Dot_TPB64 << < grid, block >> > (d_res, d_A, d_B, n);

  float h_res;
  hipMemcpy(&h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_res);
  return h_res;
}

// ------------------------------------------------------------------

__global__
void dfm2::cuda::kernel_MatMat_TPB16(
    float *C,
    const float *A,
    const float *B,
    unsigned int N)
{
  unsigned int r = blockDim.y * blockIdx.y + threadIdx.y;
  unsigned int c = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int BLOCK = 16;
  assert(blockDim.x == BLOCK);
  assert(blockDim.y == BLOCK);
  __shared__ float s_A[BLOCK][BLOCK];
  __shared__ float s_B[BLOCK][BLOCK];
  float tmp = 0.0;
  for(int i=0;i<N;i+=BLOCK){
    s_A[threadIdx.y][threadIdx.x] = A[N*r+i+threadIdx.x];
    s_B[threadIdx.y][threadIdx.x] = B[N*(i+threadIdx.y) + c];
    __syncthreads(); // wait for copy is finished for all the thread in the block
    for(int j=0;j<BLOCK;++j) {
      tmp += s_A[threadIdx.y][j] * s_B[j][threadIdx.x];
    }
    __syncthreads();
  }
  C[N*r+c] = tmp;
}

void dfm2::cuda::cuda_MatMat(
    float *h_C_gpu,
    const float *h_A,
    const float *h_B,
    unsigned int WIDTH)
{
  float *d_A, *d_B, *d_C;
  hipMalloc((void **) &d_A, sizeof(float) * WIDTH * WIDTH);
  hipMalloc((void **) &d_B, sizeof(float) * WIDTH * WIDTH);
  hipMalloc((void **) &d_C, sizeof(float) * WIDTH * WIDTH);

  hipMemcpy(d_A, h_A, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);

  const unsigned int BLOCK = 16;
  dim3 grid(WIDTH / BLOCK, WIDTH / BLOCK);
  dim3 block(BLOCK, BLOCK);

//    d_multiply0 << < grid, block >> > (d_C, d_A, d_B, WIDTH);
  kernel_MatMat_TPB16 << < grid, block >> > (d_C, d_A, d_B, WIDTH);

  hipMemcpy(h_C_gpu,
             d_C, sizeof(float) * WIDTH * WIDTH, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

// ------------------------------------------------------------------------

__global__
void dfm2::cuda::kernel_MinMax_TPB256(
    float *d_minmax,
    const float *d_XYZ,
    unsigned int np)
{
  const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int s_idx = threadIdx.x;
  assert( blockDim.y == 3 && blockIdx.y == 0 );
  unsigned int idy = threadIdx.y;
  if( idx >= np ){ return; }
  // ---------------
  const unsigned int BLOCK = 256;
  assert(blockDim.x == BLOCK);
  __shared__ float s_XYZ[BLOCK][3];
  s_XYZ[s_idx][idy] = d_XYZ[idx*3+idy];
  __syncthreads();
  if( s_idx == 0 ) {
    float vmin = s_XYZ[0][idy];
    float vmax = s_XYZ[0][idy];
    int ns = BLOCK;
    if( blockDim.x * (blockIdx.x+1) > np ) {
      ns = np - blockDim.x * blockIdx.x;
    }
    for(int is=0;is<ns;++is){
      if( s_XYZ[is][idy] < vmin ){ vmin = s_XYZ[is][idy]; }
      if( s_XYZ[is][idy] > vmax ){ vmax = s_XYZ[is][idy]; }
    }
    atomicMinFloat(d_minmax+idy+0,vmin);
    atomicMaxFloat(d_minmax+idy+3,vmax);
  }
}

void dfm2::cuda::cuda_MinMax_Point3D(
    float *h_minmax,
    const float *h_XYZ,
    unsigned int np)
{
  h_minmax[0] = h_minmax[3] = h_XYZ[0];
  h_minmax[1] = h_minmax[4] = h_XYZ[1];
  h_minmax[2] = h_minmax[5] = h_XYZ[2];
  // --------------------------------------
  float *d_minmax, *d_XYZ;
  hipMalloc((void **) &d_minmax, sizeof(float) * 6);
  hipMalloc((void **) &d_XYZ, sizeof(float) * np * 3);
  hipMemcpy(d_XYZ,
             h_XYZ, sizeof(float) * np * 3, hipMemcpyHostToDevice);
  hipMemcpy(d_minmax,
             h_minmax, sizeof(float) * 6, hipMemcpyHostToDevice);

  const unsigned int BLOCK = 256;
  dim3 grid(np/BLOCK+1);
  dim3 block(BLOCK, 3);

  kernel_MinMax_TPB256 <<< grid, block >>> (d_minmax, d_XYZ, np);

  hipMemcpy(h_minmax,
             d_minmax, sizeof(float) * 6, hipMemcpyDeviceToHost);

  hipFree(d_minmax);
  hipFree(d_XYZ);
}

