#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include "hip/hip_runtime.h"

#include "cu_matvec.h"

namespace dfm2 = delfem2;

// -------------------------------------------------------------------

__device__
void atomicMaxFloat(float * const address, const float value)
{
  if ( *address >= value ) { return; }
  int * const address_as_i = (int *)address;
  int old = * address_as_i, assumed;
  do {
    assumed = old;
    if (__int_as_float(assumed) >= value) { break; }
    old = atomicCAS(address_as_i, assumed, __float_as_int(value));
  } while (assumed != old);
}

__device__
void atomicMinFloat(float * const address, const float value)
{
  if ( *address <= value ) { return; }
  int * const address_as_i = (int *)address;
  int old = * address_as_i, assumed;
  do {
    assumed = old;
    if (__int_as_float(assumed) <= value) { break; }
    old = atomicCAS(address_as_i, assumed, __float_as_int(value));
  } while (assumed != old);
}


// -------------------------------------------------------------------

__global__
void kernel_VecScale(
    float *out,
    const float *in,
    float scale,
    const int n)
{
  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) { return; }
  out[i] = in[i] * scale;
}

void dfm2::cuda::cuda_VecScale(
    float *hOut,
    const float *hIn,
    float scale,
    const int n)
{
  float *dOut; hipMalloc((void**)&dOut, sizeof(float)*n);
  float *dIn;  hipMalloc((void**)&dIn,  sizeof(float)*n);
  hipMemcpy(dIn, hIn, sizeof(float)*n, hipMemcpyHostToDevice);

  const unsigned int tpb = 64;
  const unsigned int nblk = (unsigned int)((n-1)/tpb+1);
  kernel_VecScale<<<nblk, tpb>>>(dOut, dIn, scale, n);
  hipDeviceSynchronize();

  hipMemcpy(hOut, dOut, n * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(dOut);
  hipFree(dIn);
}


// -------------------------------------------------------------------

/**
 * @brief dot product of two vectors
 */
__global__
void kernel_Dot_TPB64(
    float* d_res,
    const float* d_A,
    const float* d_B,
    int n)
{
  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if( idx >= n ){ return; }

  const int s_idx = threadIdx.x;

  const unsigned int TPB = 64;
  __shared__ float s_prod[TPB];
  s_prod[s_idx] = d_A[idx]*d_B[idx];
  __syncthreads();

  if( s_idx == 0 ) {
    int ns = TPB;
    if( blockDim.x * (blockIdx.x+1) > n ) {
      ns = n - blockDim.x * blockIdx.x;
    }
    float blockSum = 0;
    for(int j=0;j<ns;++j){
      blockSum += s_prod[j];
    }
    atomicAdd(d_res, blockSum);
  }
}


float dfm2::cuda::cuda_Dot(
    const float* h_A,
    const float* h_B,
    unsigned int n)
{
  float *d_A, *d_B, *d_res;
  hipMalloc((void **) &d_A, sizeof(float) * n);
  hipMalloc((void **) &d_B, sizeof(float) * n);
  hipMalloc((void **) &d_res, sizeof(float));

  hipMemset((void **) &d_res, 0.f, sizeof(float));
  hipMemcpy(d_A, h_A, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, sizeof(float) * n, hipMemcpyHostToDevice);

  const unsigned int BLOCK = 64;
  dim3 grid( (n-1)/BLOCK + 1);
  dim3 block(BLOCK);

  kernel_Dot_TPB64 << < grid, block >> > (d_res, d_A, d_B, n);

  float h_res;
  hipMemcpy(&h_res, d_res, sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_res);
  return h_res;
}

// ------------------------------------------------------------------

__global__
void kernel_MatMat_TPB16(
    float *C,
    const float *A,
    const float *B,
    unsigned int N)
{
  const unsigned int r = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned int c = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int BLOCK = 16;
  assert(blockDim.x == BLOCK && blockDim.y == BLOCK);
  __shared__ float s_A[BLOCK][BLOCK];
  __shared__ float s_B[BLOCK][BLOCK];
  float tmp = 0.0;
  for(int i=0;i<N;i+=BLOCK){
    if( i+threadIdx.x < N && r < N ) {
      s_A[threadIdx.y][threadIdx.x] = A[N * r + i + threadIdx.x];
    }
    if( i+threadIdx.y < N && c < N ) {
      s_B[threadIdx.y][threadIdx.x] = B[N * (i + threadIdx.y) + c];
    }
    __syncthreads(); // wait for copy is finished for all the thread in the block
    int ns = BLOCK;
    if( i+BLOCK >= N ) { ns = N-i; }
    for(int j=0;j<ns;++j) {
      tmp += s_A[threadIdx.y][j] * s_B[j][threadIdx.x];
    }
    __syncthreads();
  }
  if( r >= N || c >= N ){ return; }
  C[N*r+c] = tmp;
}

void dfm2::cuda::cuda_MatMat(
    float *h_C_gpu,
    const float *h_A,
    const float *h_B,
    unsigned int WIDTH)
{
  float *d_A, *d_B, *d_C;
  hipMalloc((void **) &d_A, sizeof(float) * WIDTH * WIDTH);
  hipMalloc((void **) &d_B, sizeof(float) * WIDTH * WIDTH);
  hipMalloc((void **) &d_C, sizeof(float) * WIDTH * WIDTH);

  hipMemcpy(d_A, h_A, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);

  const unsigned int BLOCK = 16;
  dim3 grid( (WIDTH-1)/BLOCK+1, (WIDTH-1)/BLOCK+1);
  dim3 block(BLOCK, BLOCK);

  kernel_MatMat_TPB16 << < grid, block >> > (d_C, d_A, d_B, WIDTH);

  hipMemcpy(h_C_gpu,
             d_C, sizeof(float) * WIDTH * WIDTH, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

// ------------------------------------------------------------------------

__global__
void kernel_MinMax_TPB256(
    float *d_minmax,
    const float *d_XYZ,
    unsigned int np)
{
  const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int s_idx = threadIdx.x;
  assert( blockDim.y == 3 && blockIdx.y == 0 );
  unsigned int idy = threadIdx.y;
  if( idx >= np ){ return; }
  // ---------------
  const unsigned int BLOCK = 256;
  assert(blockDim.x == BLOCK);
  __shared__ float s_XYZ[BLOCK][3];
  s_XYZ[s_idx][idy] = d_XYZ[idx*3+idy];
  __syncthreads();
  if( s_idx == 0 ) {
    float vmin = s_XYZ[0][idy];
    float vmax = s_XYZ[0][idy];
    int ns = BLOCK;
    if( blockDim.x * (blockIdx.x+1) > np ) {
      ns = np - blockDim.x * blockIdx.x;
    }
    for(int is=0;is<ns;++is){
      if( s_XYZ[is][idy] < vmin ){ vmin = s_XYZ[is][idy]; }
      if( s_XYZ[is][idy] > vmax ){ vmax = s_XYZ[is][idy]; }
    }
    atomicMinFloat(d_minmax+idy+0,vmin);
    atomicMaxFloat(d_minmax+idy+3,vmax);
  }
}

void dfm2::cuda::cuda_MinMax_Point3D(
    float *h_minmax,
    const float *h_XYZ,
    unsigned int np)
{
  h_minmax[0] = h_minmax[3] = h_XYZ[0];
  h_minmax[1] = h_minmax[4] = h_XYZ[1];
  h_minmax[2] = h_minmax[5] = h_XYZ[2];
  // --------------------------------------
  float *d_minmax, *d_XYZ;
  hipMalloc((void **) &d_minmax, sizeof(float) * 6);
  hipMalloc((void **) &d_XYZ, sizeof(float) * np * 3);
  hipMemcpy(d_XYZ,
             h_XYZ, sizeof(float) * np * 3, hipMemcpyHostToDevice);
  hipMemcpy(d_minmax,
             h_minmax, sizeof(float) * 6, hipMemcpyHostToDevice);

  {
    const unsigned int BLOCK = 256;
    dim3 grid((np - 1) / BLOCK + 1);
    dim3 block(BLOCK, 3);
    kernel_MinMax_TPB256 <<< grid, block >>> (d_minmax, d_XYZ, np);
  }

  hipMemcpy(h_minmax,
             d_minmax, sizeof(float) * 6, hipMemcpyDeviceToHost);

  hipFree(d_minmax);
  hipFree(d_XYZ);
}

// ---------------------------------------------------------------------------------------------------------------------

__device__
void kernel_dist3(
    float *d,
    const float p0[3],
    const float p1[3])
{
  float v = (p1[0]-p0[0])*(p1[0]-p0[0]) + (p1[1]-p0[1])*(p1[1]-p0[1]) + (p1[2]-p0[2])*(p1[2]-p0[2]);
  *d = sqrtf(v);
}

__global__
void kernel_CentRad_MeshTri3D_TPB256(
    float *dXYZ_c,
    float *dRad,
    const float *dXYZ,
    const unsigned int nXYZ,
    const unsigned int *dTri,
    const unsigned int nTri)
{
  const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if( idx >= nTri ) return;
  // ----------------------------
  const unsigned int itri = idx;
  const unsigned int i0 = dTri[itri*3+0];
  const unsigned int i1 = dTri[itri*3+1];
  const unsigned int i2 = dTri[itri*3+2];
  const float p0[3] = {dXYZ[i0*3+0],dXYZ[i0*3+1],dXYZ[i0*3+2]};
  const float p1[3] = {dXYZ[i1*3+0],dXYZ[i1*3+1],dXYZ[i1*3+2]};
  const float p2[3] = {dXYZ[i2*3+0],dXYZ[i2*3+1],dXYZ[i2*3+2]};
  const float pc[3] = {
      (p0[0]+p1[0]+p2[0])/3.f,
      (p0[1]+p1[1]+p2[1])/3.f,
      (p0[2]+p1[2]+p2[2])/3.f };
  dXYZ_c[itri*3+0] = pc[0];
  dXYZ_c[itri*3+1] = pc[1];
  dXYZ_c[itri*3+2] = pc[2];
  // ---------------------
  float l0,l1,l2;
  kernel_dist3(&l0, pc, p0);
  kernel_dist3(&l1, pc, p1);
  kernel_dist3(&l2, pc, p2);
  if( l0 > l1 && l0 > l2 ){ dRad[itri] = l0; return; }
  if( l1 > l0 && l1 > l2 ){ dRad[itri] = l1; return; }
  dRad[itri] = l2;
}

void dfm2::cuda::cuda_CentRad_MeshTri3D(
    float* hXYZ_c,
    float* hRad,
    const float *hXYZ,
    const unsigned int nXYZ,
    const unsigned int *hTri,
    const unsigned int nTri)
{
  float *dXYZ, *dXYZ_c, *dRad;
  unsigned int *dTri;
  hipMalloc((void **) &dXYZ, sizeof(float) * nXYZ * 3);
  hipMalloc((void **) &dTri, sizeof(unsigned int) * nTri * 3);
  hipMalloc((void **) &dXYZ_c, sizeof(float) * nTri * 3);
  hipMalloc((void **) &dRad, sizeof(float) * nTri);
  hipMemcpy(dXYZ,
             hXYZ, sizeof(float) * nXYZ * 3, hipMemcpyHostToDevice);
  hipMemcpy(dTri,
             hTri, sizeof(unsigned int) * nTri * 3, hipMemcpyHostToDevice);

  {
    const unsigned int BLOCK = 64;
    dim3 grid( (nTri-1)/BLOCK + 1 );
    dim3 block( BLOCK );
    kernel_CentRad_MeshTri3D_TPB256 <<< grid, block >>> (dXYZ_c, dRad,
        dXYZ, nXYZ,
        dTri, nTri);
  }

  hipMemcpy(hXYZ_c,
             dXYZ_c, sizeof(float) * nTri * 3, hipMemcpyDeviceToHost);
  hipMemcpy(hRad,
             dRad, sizeof(float) * nTri, hipMemcpyDeviceToHost);

  hipFree(dTri);
  hipFree(dXYZ);
  hipFree(dXYZ_c);
  hipFree(dRad);
}
